#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/
#include "CudaKernels.h"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions


texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
texture<uchar4, 2, hipReadModeElementType> uchar4Tex;
texture<float, 2, hipReadModeElementType> floatTex;
texture<uchar, 2> ucharTex;



__device__ uint rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(fabs(rgba.y));
	rgba.z = __saturatef(fabs(rgba.z));
	rgba.w = __saturatef(fabs(rgba.w));
	return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}


__device__ float4 rgbaIntToFloat(uint c)
{
	float4 rgba;
	rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
	rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;  //  /255.0f;
	rgba.z = ((c >> 16) & 0xff) * 0.003921568627f; //  /255.0f;
	rgba.w = ((c >> 24) & 0xff) * 0.003921568627f; //  /255.0f;
	return rgba;
}


__global__ void
d_passthrough_texture_uint(uint* pImage, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}

	float4 pixel = tex2D(rgbaTex, x, y);
	pImage[y * w + x] = rgbaFloatToInt(pixel);
	
	return;
}



__global__ void
d_invert_pixel_uint(uint* pImage, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}

	float4 pixel = tex2D(rgbaTex, x, y);
	float4 pixel_inverted;
	pixel_inverted.x = 1.0f - pixel.z;
	pixel_inverted.y = 1.0f - pixel.y;
	pixel_inverted.z = 1.0f - pixel.x;
	pImage[y * w + x] = rgbaFloatToInt(pixel_inverted);

	return;
}


__global__ void
d_passthrough_texture_uchar(uchar* pImage, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}

	uchar pixel = tex2D(ucharTex, x, y);
	pImage[y * w + x] = pixel;

	return;
}




__global__ void
d_invert_pixel_uchar(uchar* pImage, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}

	uchar pixel = tex2D(ucharTex, x, y);
	pImage[y * w + x] = 255 - pixel;

	return;
}



__global__ void
d_convert_rgba_to_gray(uchar* pOutImage, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}

	uchar4 pixel = tex2D(uchar4Tex, x, y);
	float color = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;
	pOutImage[y * w + x] = (uchar)color;

	return;
}




extern "C"
{
	void passthrough_texture_uint(uint* dOutputImage, uint* dInputImage, int width, int height, size_t pitch, bool invert_channel)
	{
		// Bind the array to the texture
		hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
		checkCudaErrors(hipBindTexture2D(0, rgbaTex, dInputImage, desc, width, height, pitch));

		const dim3 threads_per_block(16, 16);
		dim3 num_blocks;
		num_blocks.x = (width + threads_per_block.x - 1) / threads_per_block.x;
		num_blocks.y = (height + threads_per_block.y - 1) / threads_per_block.y;

		if (invert_channel)
			d_invert_pixel_uint << <  num_blocks, threads_per_block >> >(dOutputImage, width, height);
		else
			d_passthrough_texture_uint << <  num_blocks, threads_per_block >> >(dOutputImage, width, height);
	}



	void passthrough_texture_uchar(uchar* dOutputImage, uchar* dInputImage, int width, int height, size_t pitch, bool invert_channel)
	{
		// Bind the array to the texture
		hipChannelFormatDesc desc = hipCreateChannelDesc<uchar>();
		checkCudaErrors(hipBindTexture2D(0, ucharTex, dInputImage, desc, width, height, pitch));

		const dim3 threads_per_block(16, 16);
		dim3 num_blocks;
		num_blocks.x = (width + threads_per_block.x - 1) / threads_per_block.x;
		num_blocks.y = (height + threads_per_block.y - 1) / threads_per_block.y;

		if (invert_channel)
			d_invert_pixel_uchar << <  num_blocks, threads_per_block >> >(dOutputImage, width, height);
		else
			d_passthrough_texture_uchar << <  num_blocks, threads_per_block >> >(dOutputImage, width, height);
	}



	void convert_rgba_to_gray(uchar4* dInputImage, int width, int height, size_t input_pitch, uchar* dOutputImage)
	{
		// Bind the array to the texture
		hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
		checkCudaErrors(hipBindTexture2D(0, uchar4Tex, dInputImage, desc, width, height, input_pitch));

		const dim3 threads_per_block(16, 16);
		dim3 num_blocks;
		num_blocks.x = (width + threads_per_block.x - 1) / threads_per_block.x;
		num_blocks.y = (height + threads_per_block.y - 1) / threads_per_block.y;

		d_convert_rgba_to_gray << <  num_blocks, threads_per_block >> >(dOutputImage, width, height);
	}


	
}; // extern "C"