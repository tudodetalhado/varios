#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include "CudaKernels.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

//typedef unsigned char VolumeType;
typedef float2 VolumeType;

//texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<VolumeType, 3, hipReadModeElementType>		volumeTex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         volumeTransferTex; // 1D transfer function texture

typedef struct
{
	float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
	float3 o;   // origin
	float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.d;
	float3 tbot = invR * (boxmin - r.o);
	float3 ttop = invR * (boxmax - r.o);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}

__device__ uint volrend_rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(rgba.y);
	rgba.z = __saturatef(rgba.z);
	rgba.w = __saturatef(rgba.w);
	return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) | (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__global__ void
d_render(uint *d_output, uint imageW, uint imageH,
float density, float brightness,
float transferOffset, float transferScale)
{
	const int maxSteps = 500;
	const float tstep = 0.01f;
	const float opacityThreshold = 0.95f;
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	eyeRay.d = normalize(make_float3(u, v, -2.0f));
	eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) return;

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

	// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float last_tsdf = tex3D(volumeTex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f).x;

	for (int i = 0; i<maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float2 sample = tex3D(volumeTex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);

		//sample *= 64.0f;    // scale for 10-bit data

#if 0
		// lookup in transfer function texture
		float4 col = tex1D(volumeTransferTex, (sample - transferOffset)*transferScale);
		col.w *= density;

		// "under" operator for back-to-front blending
		//sum = lerp(sum, col, col.w);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;
		// "over" operator for front-to-back blending
		sum = sum + col*(1.0f - sum.w);

		// exit early if opaque
		if (sum.w > opacityThreshold)
			break;
#else
		//if (sample.x > 1)
		//{
		//	sum = make_float4(1);
		//	break;
		//}

		float tsdf = sample.x;
		if (std::signbit(tsdf) != std::signbit(last_tsdf))
		{
			sum = make_float4(1);
			break;
		}
		else
		{
			last_tsdf = tsdf;
		}

#endif
		t += tstep;

		if (t > tfar) break;

		pos += step;
	}

	sum *= brightness;

	// write output color
	d_output[y*imageW + x] = volrend_rgbaFloatToInt(sum);
}

extern "C"
void setTextureFilterMode(bool bLinearFilter)
{
	volumeTex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void volume_render_init(void *h_volume, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	volumeTex.normalized = true;                      // access with normalized texture coordinates
	volumeTex.filterMode = hipFilterModeLinear;      // linear interpolation
	volumeTex.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	volumeTex.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(volumeTex, d_volumeArray, channelDesc));

	// create transfer function texture
	float4 transferFunc[] =
	{
#if 0
		{ 0.0, 0.0, 0.0, 0.0, },
		{ 1.0, 0.0, 0.0, 1.0, },
		{ 1.0, 0.5, 0.0, 1.0, },
		{ 1.0, 1.0, 0.0, 1.0, },
		{ 0.0, 1.0, 0.0, 1.0, },
		{ 0.0, 1.0, 1.0, 1.0, },
		{ 0.0, 0.0, 1.0, 1.0, },
		{ 1.0, 0.0, 1.0, 1.0, },
		{ 0.0, 0.0, 0.0, 0.0, },
#else
#if 0
		{ 0.0, 0.0, 0.0, 0.0, },
		{ 0.0, 0.0, 1.0, 0.1, },
		{ 0.0, 0.3, 1.0, 0.2, },
		{ 0.0, 0.7, 1.0, 0.3, },
		{ 0.1, 1.0, 0.8, 0.4, },
		{ 0.5, 1.0, 0.5, 0.5, },
		{ 0.8, 1.0, 0.1, 0.6, },
		{ 1.0, 0.8, 0.0, 0.7, },
		{ 1.0, 0.4, 0.0, 0.8, },
		{ 0.8, 0.0, 0.0, 0.9, },
		{ 0.5, 0.0, 0.0, 1.0, },
#else
		{ 0.0, 0.0, 0.0, 0.0, },
		{ 1.0, 1.0, 1.0, 1.0, },
#endif
#endif
	};

	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	hipArray *d_transferFuncArray;
	checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

	volumeTransferTex.filterMode = hipFilterModeLinear;
	volumeTransferTex.normalized = true;    // access with normalized texture coordinates
	volumeTransferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(volumeTransferTex, d_transferFuncArray, channelDesc2));
}

extern "C"
void volume_render_cleanup()
{
	checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH,
float density, float brightness, float transferOffset, float transferScale)
{
	d_render << <gridSize, blockSize >> >(d_output, imageW, imageH, density,
		brightness, transferOffset, transferScale);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}


#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
