#include "hip/hip_runtime.h"
#include "CudaKernels.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <thrust/device_vector.h>


__device__ uint rgba_float_to_int(float4 rgba)
{
	rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(fabs(rgba.y));
	rgba.z = __saturatef(fabs(rgba.z));
	rgba.w = __saturatef(fabs(rgba.w));
	return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}

struct Ray
{
	float3 origin;   // origin
	float3 direction;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm
__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.direction;
	float3 tbot = invR * (boxmin - r.origin);
	float3 ttop = invR * (boxmax - r.origin);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}


typedef struct
{
	float4 m[3];
} float3x4;
__constant__ float3x4 camera_to_world_dev_matrix;  // inverse view matrix


// transform vector by matrix (no translation)
__device__ float3 mul(const float3x4 &M, const float3 &v)
{
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

// transform vector by matrix with translation
__device__ float4 mul(const float3x4 &M, const float4 &v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}

extern "C"
{

	__global__ void update_vb(float *d_verts_ptr, int vertex_count, float timeElapsed)
	{
		const unsigned long long int threadId = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadId < vertex_count * 4)
		{
			float valx = d_verts_ptr[threadId * 4 + 0];
			float valy = d_verts_ptr[threadId * 4 + 1];
			float valz = d_verts_ptr[threadId * 4 + 2];


			d_verts_ptr[threadId * 4 + 0] = valx * timeElapsed;
			d_verts_ptr[threadId * 4 + 1] = valy * timeElapsed;
			d_verts_ptr[threadId * 4 + 2] = valz * timeElapsed;
		}
	}

	void cuda_kernel(float *d_verts_ptr, int vertex_count, float timeElapsed)
	{
		if (vertex_count > 1024)
			update_vb << <vertex_count / 1024 + 1, 1024 >> >(d_verts_ptr, vertex_count, timeElapsed);
		else
			update_vb << <1, vertex_count >> >(d_verts_ptr, vertex_count, timeElapsed);
	}






	__global__ void	raycast_box_kernel(
		uint* out_image,
		ushort image_width,
		ushort image_height,
		ushort3 box_size)
	{
		ulong x = blockIdx.x * blockDim.x + threadIdx.x;
		ulong y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= image_width || y >= image_height)
			return;

		// Convert from image space (in pixels) to screen space
		float u = (x / (float)image_width) * 2.0f - 1.0f;
		float v = (y / (float)image_height) * 2.0f - 1.0f;
		Ray eye_ray;
		eye_ray.origin = make_float3(mul(camera_to_world_dev_matrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
		float3 screen_coord = normalize(make_float3(u, -v, -2.0f));
		eye_ray.direction = mul(camera_to_world_dev_matrix, screen_coord);

		// find intersection with box
		const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
		const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);
		float tnear, tfar;
		
		if (intersectBox(eye_ray, boxMin, boxMax, &tnear, &tfar))
			out_image[y * image_width + x] = rgba_float_to_int(make_float4(0.f, 0.5f, 0.5f, 1.f));
		else
			out_image[y * image_width + x] = rgba_float_to_int(make_float4(0.f, 0.0f, 0.0f, 1.f));
	}

	void raycast_box(
		uint* image_data_ref,
		ushort width,
		ushort height,
		float* camera_to_world_mat3x4,
		ushort3 box_size)
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(camera_to_world_dev_matrix), camera_to_world_mat3x4, sizeof(float4) * 3));

		const dim3 threads_per_block(16, 16);
		dim3 num_blocks = dim3(iDivUp(width, threads_per_block.x), iDivUp(height, threads_per_block.y));;

		raycast_box_kernel << < num_blocks, threads_per_block >> >(
			image_data_ref,
			width,
			height,
			box_size
			);

		checkCudaErrors(hipDeviceSynchronize());
	}

};